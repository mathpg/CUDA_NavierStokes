
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>

#define N 1000
#define MIN(x,y) (x<y ? x:y)
#define SQR(x) (x*x)

#define REDCT_NUMTHREADS 256

#define NUMBLOCKS 32

//Spacial Data
double xlen; //
double ylen;
int imax;
int jmax;
double delx;
double dely;

//Time Data
double ttime=0;
double final_time;
double del_time;
double tau; //factor for time step control

//Pressure Data
int max_iter; //max numer of presssure iterations for a time step
int iter; // SOR iter counter
double res; //norm of pressure equation residual
double eps; //stopping tolerance eps for pressure iteration
double omg; //relaxation parameter u> for SOR iteration
double gam; //upwind differencing factor 


//Problem dependent Data
double Re;
double gx;
double gy;
double vx_init;
double vy_init;
double p_init;
int wW,wE,wN,wS; /*specify the type of boundary condition along the
					western (left), eastern (right), northern (upper), and
					southern (lower) boundaries of 17 = [0,xlength] x
					[0,ylength]; each may have one of the values:
					1 for free-slip conditions,
					2 for no-slip conditions,*/
char problem[N];
struct timeval start, end;
int state=0;
int n_iter=5;

double absf(double a){
	return a < 0? -1*a : a;
}

void print_values(){
	printf("Printing values:\n");
	printf("xlen=%f\n",xlen);
	printf("ylen=%f\n",ylen);
	printf("imax=%d\n",imax);
	printf("jmax=%d\n",jmax);
	printf("delx=%f\n",delx);
	printf("dely=%f\n",dely);
	printf("final_time=%f\n",final_time);
	printf("del_time=%f\n",del_time);
	printf("tau=%f\n",tau);
	printf("max_iter=%d\n",max_iter);
	printf("res=%f\n",res);
	printf("eps=%f\n",eps);
	printf("omg=%f\n",omg);
	printf("gam=%f\n",gam);
	printf("Re=%f\n",Re);
	printf("gx=%f\n",gx);
	printf("gy=%f\n",gy);
	printf("vx_init=%f\n",vx_init);
	printf("vy_init=%f\n",vy_init);
	printf("p_init=%f\n",p_init);
	printf("wW=%d\n",wW);
	printf("wE=%d\n",wE);
	printf("wN=%d\n",wW);
	printf("wS=%d\n",wS);
	printf("problem=%s\n",problem);
	printf("-------------------------------\n\n");
}

void read_file(char * file_name){
	FILE *fp;
	char ch[50];
	char* s,*e;
	int num_param=0;
	fp = fopen(file_name, "r"); // read mode

	if (fp == NULL){
	  perror("Error while opening the file.\n");
	  exit(EXIT_FAILURE);
	}
	fscanf(fp,"%s", ch);
	while( !feof(fp)){ 
		s=strtok(ch,":");
		switch(num_param){
			case 0:
				xlen=strtod(s,&e);
				break;
			case 1:
				ylen=strtod(s,&e);
				break;
			case 2:
				imax=strtod(s,&e);
				break;
			case 3:
				jmax=strtod(s,&e);
				break;
			case 4:
				delx=strtod(s,&e);
				if(delx==0){
					delx=xlen/imax;
				}
				break;
			case 5:
				dely=strtod(s,&e);
				if(dely==0){
					dely=ylen/jmax;
				}
				break;
			case 6:
				final_time=strtod(s,&e);
				break;
			case 7:
				del_time=strtod(s,&e);
				break;
			case 8:
				tau=strtod(s,&e);
				break;
			case 9:
				max_iter=strtod(s,&e);
				break;
			case 10:
				res=strtod(s,&e);
				break;
			case 11:
				eps=strtod(s,&e);
				break;
			case 12:
				omg=strtod(s,&e);
				break;
			case 13:
				gam=strtod(s,&e);
				break;
			case 14:
				Re=strtod(s,&e);
				break;
			case 15:
				gx=strtod(s,&e);
				break;
			case 16:
				gy=strtod(s,&e);
				break;
			case 17:
				vx_init=strtod(s,&e);
				break;
			case 18:
				vy_init=strtod(s,&e);
				break;
			case 19:
				p_init=strtod(s,&e);
				break;
			case 20:
				wW=strtod(s,&e);
				break;
			case 21:
				wE=strtod(s,&e);
				break;
			case 22:
				wN=strtod(s,&e);
				break;
			case 23:
				wS=strtod(s,&e);
				break;
			case 24:
				strcpy(problem,s);
				break;	
			default:			
				exit(666);
				break;
			}
		num_param++;
		fscanf(fp,"%s", ch);
	}
	fclose(fp);
}




double *vx;
double *vy;
double *p;
double *rhs;
double *F;
double *G;

double *d_vx = NULL;
double *d_vy = NULL;
double *d_p = NULL;
double *d_p_prev = NULL;
double *d_partial = NULL;
double *d_p_diff = NULL;

double *d_rhs = NULL;
double *d_F = NULL;
double *d_G = NULL;
double *d_flag = NULL;

double *d_diag_n = NULL;
double *d_diag_s = NULL;
double *d_diag_w = NULL;
double *d_diag_e = NULL;
double *d_diag_p = NULL; 

double *d_maxdiff = NULL;

double *d_vxdiff = NULL;
double *d_vydiff = NULL;
double *d_vxflag = NULL;
double *d_vyflag = NULL;

int *d_res = NULL;

double *dp=NULL;
	

int n_threads,n_blocos ;

hipError_t err = hipSuccess;

void write_file(char * output){
	FILE *fp;
	int i,j,idx;
	fp = fopen(output, "w"); // write mode

	if (fp == NULL){
	  perror("Error while opening the file.\n");
	  exit(EXIT_FAILURE);
	}
	fprintf(fp,"Time taken: %ld seconds\n", end.tv_sec-start.tv_sec);
	fprintf(fp,"Simulation Time: %.5f seconds\n", ttime);
	for(i=0;i<imax+2;i++){
		for(j=0;j<jmax+2;j++){
			idx=i*(imax+2)+j;
			fprintf(fp,"vx[%d][%d]=%.10f\n",i,j,vx[idx]);
			fprintf(fp,"vy[%d][%d]=%.10f\n",i,j,vy[idx]);
			fprintf(fp,"F[%d][%d]=%.10f\n",i,j,F[idx]);
			fprintf(fp,"G[%d][%d]=%.10f\n",i,j,G[idx]);
			fprintf(fp,"p[%d][%d]=%.10f\n",i,j,p[idx]);
		}
		//fprintf(fp,"\n");
	}
	fclose(fp);
}

void alocate_vectors_host(){
	vx = (double *)malloc((imax+2)*(jmax+2) * sizeof(double));
	vy = (double *)malloc((imax+2)*(jmax+2) * sizeof(double));
	p = (double *)malloc((imax+2)*(jmax+2) * sizeof(double));
	rhs = (double *)malloc((imax+2)*(jmax+2) * sizeof(double));
	F = (double *)malloc((imax+2)*(jmax+2) * sizeof(double));
	G = (double *)malloc((imax+2)*(jmax+2) * sizeof(double));
	
	if(vx==NULL || vy==NULL || p==NULL || rhs==NULL || F==NULL || G==NULL){
		printf("It wasn't possible to alocate memory\n");
		exit(0);
	}
}

void alocate_vectors_device(){
	
	size_t size = (imax+2)*(jmax+2) * sizeof(double);
	
	err = hipMalloc((void **)&d_vx, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector vx (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	
	err = hipMalloc((void **)&d_vy, size);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector vy (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	
	err = hipMalloc((void **)&d_p, size);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector p (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMalloc((void **)&d_p_prev, size);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device previous vector p (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMalloc((void **)&d_p_diff, size);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device diffential vector p (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	
	err = hipMalloc((void **)&d_partial, NUMBLOCKS * sizeof(double));
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device partial vector (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	
	err = hipMalloc((void **)&d_rhs, size);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector rhs (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	
	err = hipMalloc((void **)&d_F, size);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector F (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	
	err = hipMalloc((void **)&d_G, size);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector G (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMalloc((void **)&d_diag_n, size);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector diag_n (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
	err = hipMalloc((void **)&d_diag_s, size);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector diag_s (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMalloc((void **)&d_diag_w, size);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector diag_w (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMalloc((void **)&d_diag_e, size);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector diag_e (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMalloc((void **)&d_diag_p, size);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector diag_p (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMalloc((void **)&d_flag, size);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_flag (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	err = hipMalloc((void **)&d_vxflag, size);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_vxflag (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **)&d_vyflag, size);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_vyflag (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	   
    err= hipMalloc((void **)&d_res,sizeof(int));
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device pointer res (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err= hipMalloc((void **)&d_maxdiff,sizeof(double));
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device pointer maxdiff (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err= hipMalloc((void **)&d_vxdiff,size);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device pointer vxdiff (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err= hipMalloc((void **)&d_vydiff,size);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device pointer vydiff (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void free_vectors_host(){
	free(vx);
	free(vy);
	free(p);
	free(rhs);
	free(F);
	free(G);	
}

void free_vectors_device(){
	
	err = hipFree(d_vx);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector vx (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipFree(d_vy);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector vy (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipFree(d_p);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector p (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipFree(d_p_prev);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device previous vector p (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipFree(d_rhs);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector rhs (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipFree(d_F);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector F (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipFree(d_G);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector G (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipFree(d_res);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device pointer res (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    
    err = hipFree(d_maxdiff);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device pointer maxdiff (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
     err = hipFree(d_vxdiff);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device pointer vxdiff (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
     err = hipFree(d_vydiff);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device pointer vydiff (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    	
}

void copy_vectors_host_to_device(){
	size_t size = (imax+2)*(jmax+2) * sizeof(double);
	
	err = hipMemcpy(d_vx, vx, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector vx from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMemcpy(d_vy, vy, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector vy from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMemcpy(d_p, p, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector p from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMemcpy(d_rhs, rhs, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector rhs from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMemcpy(d_F, F, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector F from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMemcpy(d_G, G, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector G from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }   
}

void copy_vectors_device_to_host(){
	size_t size = (imax+2)*(jmax+2) * sizeof(double);
	
	err = hipMemcpy(vx, d_vx, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector vx from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMemcpy(vy, d_vy, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector vy from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMemcpy(p, d_p, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector p from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMemcpy(rhs, d_rhs, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector rhs from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMemcpy(F, d_F, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector F from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMemcpy(G, d_G, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector G from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
      
    
}

__device__ double d_absf(double x){
	return x < 0 ? -1*x:x;
}

__global__ void init_UVP(int imax,int jmax,double vx_init,double vy_init,double p_init ,double * d_vx,double * d_vy,double * d_p,double * d_rhs,double * d_F,double * d_G){
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	
	int n = (imax+2)*(jmax+2);
	if(idx < n){
		d_vx[idx]=vx_init;
		d_vy[idx]=vy_init;
		d_p[idx]=p_init;
		d_rhs[idx]=0;
		d_F[idx]=0;
		d_G[idx]=0;
	}
}

__global__ void dt_reductionMax(int imax,int jmax, double *d_partial, double *d_v){
	
	__shared__ 	double cache [REDCT_NUMTHREADS] ;
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	double temp = 0;
	int n = (imax+2)*(jmax+2);
	int inc = blockDim.x*gridDim.x;
	int i,k;
	for(i=idx;i<n;i+=inc){
		if(i > (imax+1) &&  i < (jmax+1)*(imax+2) && i%(imax+2) > 0 && i%(imax+2) < (imax+1)){
			if(temp < d_absf(d_v[i])) {
				temp = d_absf(d_v[i]);
			}
		}
	}
	cache[threadIdx.x]=temp;
	__syncthreads();
	
	for( k =(blockDim.x >> 1);k>0;k>>=1){
		if(threadIdx.x < k){
			if(cache[threadIdx.x] < cache[threadIdx.x+k]){
				cache[threadIdx.x] = cache[threadIdx.x+k];
			}
		}
		__syncthreads();
	}	
	if(threadIdx.x==0){
		d_partial[blockIdx.x]=cache[0];
	}

}

void comp_delt(){
	int i;
	double aux;
	double aux2;
	double partialvx[NUMBLOCKS];
	double partialvy[NUMBLOCKS];
	double maxvx=0;
	double maxvy=0;
	if(tau > 0) {
		
		dt_reductionMax<<<REDCT_NUMTHREADS,NUMBLOCKS>>>(imax,jmax,d_partial,d_vx);
		err = hipMemcpy(partialvx, d_partial, NUMBLOCKS*sizeof(double), hipMemcpyDeviceToHost);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to copy pointer d_partial from device to host vx (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}
		dt_reductionMax<<<REDCT_NUMTHREADS,NUMBLOCKS>>>(imax,jmax,d_partial,d_vy);
		
		err = hipMemcpy(partialvy, d_partial, NUMBLOCKS*sizeof(double), hipMemcpyDeviceToHost);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to copy pointer d_partial from device to host vy (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}
		for(i=0;i<NUMBLOCKS;i++){	
			if(partialvx[i] > maxvx) maxvx=partialvx[i];
			if(partialvy[i] > maxvy) maxvy=partialvy[i];
		}
		aux= MIN((delx/maxvx) ,(dely/maxvy));
		aux2=MIN(aux, ((Re/2)*(SQR(delx)*SQR(dely)/(SQR(delx)+SQR(dely)))));
		del_time=tau*aux2;
	}
}

__global__ void set_NorthBond(int imax, int jmax,int wN,double *d_vx, double * d_vy){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	
	int n = (imax+2)*(jmax+2);
	switch(wN){
		case 1://free-slip condition
			if(idx < n && idx > 0 && idx < imax+1){
				d_vx[idx]=d_vx[imax+2+idx];
				d_vy[imax+2+idx]=0;
			}
			break;			
		case 2://no-slip condition
			if(idx < n && idx > 0 && idx < imax+1){
				d_vx[idx]=-d_vx[imax+2+idx];
				d_vy[imax+2+idx]=0;
			}	
			break;
		default:
			if(idx < n && idx > 0 && idx < imax+1){
				d_vx[idx]=d_vx[imax+2+idx];
				d_vy[imax+2+idx]=0;
			}
			break;
	}
}

__global__ void set_SouthBond(int imax,int jmax,int wS, double *d_vx, double *d_vy){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	
	int n = (imax+2)*(jmax+2);
	switch(wS){
		case 1: //free-slip condition
			if(idx < n && idx > 0 && idx < imax+1){
				d_vx[(jmax+1)*(imax+2)+idx]=d_vx[jmax*(imax+2)+idx];
				d_vy[(jmax)*(imax+2)+idx]=0;
			}
			break;
		case 2://no-slip condition
			if(idx < n && idx > 0 && idx < imax+1){
				d_vx[(jmax+1)*(imax+2)+idx]=-d_vx[jmax*(imax+2)+idx];
				d_vy[(jmax+1)*(imax+2)+idx]=0;
			}
			break;
		default:
			if(idx < n && idx > 0 && idx < imax+1){
				d_vx[(jmax+1)*(imax+2)+idx]=d_vx[jmax*(imax+2)+idx];
				d_vy[(jmax)*(imax+2)+idx]=0;
			}
			break;	
	}
}

__global__ void set_WestBond(int imax,int jmax,int wW, double *d_vx, double *d_vy){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	
	int n = (imax+2)*(jmax+2);
	switch(wW){
		case 1: //free-slip condition
			if(idx < n && idx%(imax+2)==1 && idx > (imax+1) && idx < (n-imax-2)){
				d_vx[idx-1]=0;
				d_vy[idx-1]=d_vy[idx];
			}
			break;
		case 2://no-slip condition
			if(idx < n && idx%(imax+2)==1 && idx > (imax+1) && idx < (n-imax-2)){
				d_vx[idx-1]=0;
				d_vy[idx-1]=-d_vy[idx];
			}			
			break;
		default:
			if(idx < n && idx%(imax+2)==1 && idx > (imax+1) && idx < (n-imax-2)){
				d_vx[idx-1]=0;
				d_vy[idx-1]=d_vy[idx];
			}
			break;
	}
}

__global__ void set_EastBond(int imax,int jmax,int wE, double *d_vx, double *d_vy){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	
	int n = (imax+2)*(jmax+2);
	switch(wE){
		case 1: //free-slip condition
			if(idx < n && idx%(imax+2)==(imax+1) && idx > (imax+1) && idx < (n-imax-2)){
				d_vx[idx-1]=0;
				d_vy[idx]=d_vy[idx-1];
			}
			break;
		case 2://no-slip condition
			if(idx < n && idx%(imax+2)==(imax+1) && idx > (imax+1) && idx < (n-imax-2)){
				d_vx[idx-1]=0;
				d_vy[idx]=-d_vy[idx-1];
			}
			break;
		default:
				if(idx < n && idx%(imax+2)==(imax+1) && idx > (imax+1) && idx < (n-imax-2)){
				d_vx[idx-1]=0;
				d_vy[idx]=d_vy[idx-1];
			}
			break;
	}
}

void set_bondCond(){
	set_NorthBond<<< n_blocos, n_threads >>>(imax,jmax,wN,d_vx,d_vy);
	set_SouthBond<<< n_blocos, n_threads >>>(imax,jmax,wS,d_vx,d_vy);
	set_WestBond<<< n_blocos, n_threads >>>(imax,jmax,wW,d_vx,d_vy);
	set_EastBond<<< n_blocos, n_threads >>>(imax,jmax,wE,d_vx,d_vy);
}

__global__ void set_lidDrivenCavityProblem(double lid_vel,int imax,int jmax, double *d_vx){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	
	int n = (imax+2)*(jmax+2);
	if(idx < n && idx > 0 && idx < imax+1){
				d_vx[idx]=2.0*lid_vel - d_vx[imax+2+idx];
	}
}

__device__ double del_vx_sqr_del_x(double gam, double delx, double vx_C_point,double vx_W_point,double vx_E_point){ //d(vx²)/dx
	double aws;
	aws=(SQR((vx_C_point+vx_E_point)/2) - SQR((vx_W_point+vx_C_point)/2))/delx;
	aws+=(gam/delx)*((d_absf(vx_C_point+vx_E_point)/2)*((vx_C_point-vx_E_point)/2) - (d_absf(vx_W_point+vx_C_point)/2)*((vx_W_point-vx_C_point)/2));
	return aws;
}

__device__ double del_vx_vy_del_y(double gam, double dely,double vx_N_point,double vx_C_point,double vx_S_point, double vy_C_point, double vy_E_point, double vy_S_point, double vy_SE_point){ //d(vx*vy)/dy
	double aws;
	aws=((vy_C_point+vy_E_point)*(vx_C_point+vx_N_point) - (vy_S_point+vy_SE_point)*(vx_S_point+vx_C_point))/2*dely;
	aws+=(gam/2*dely)*(d_absf(vy_C_point+vy_E_point)*(vx_C_point-vx_N_point) - d_absf(vy_S_point+vy_SE_point)*(vx_S_point-vx_C_point));
	return aws;
}

__device__ double del_sqr_vx_del_sqr_x(double delx,double vx_E_point, double vx_C_point, double vx_W_point){ //d²(vx)/dx²
	return (vx_E_point-2*vx_C_point+vx_W_point)/SQR(delx);
}

__device__ double del_sqr_vx_del_sqr_y(double dely,double vx_N_point, double vx_C_point, double vx_S_point){ //d²(vx)/dy²
	return (vx_N_point-2*vx_C_point+vx_S_point)/SQR(dely);
}


__device__ double del_vy_sqr_del_y(double gam, double dely,double vy_N_point, double vy_C_point, double vy_S_point){ // d(vy²)/dy
	double aws;
	aws = (SQR((vy_C_point+vy_N_point)/2) - SQR((vy_S_point+vy_C_point)/2))/dely;
	aws+=(gam/dely)*(d_absf((vy_C_point+vy_N_point)/2)*((vy_C_point-vy_N_point)/2) - d_absf((vy_S_point+vy_C_point)/2)*((vy_S_point-vy_C_point)/2));
	return aws;
}


__device__ double del_vy_vx_del_x(double gam, double delx,double vy_W_point,double vy_C_point,double vy_E_point, double vx_N_point,double vx_C_point,double vx_W_point, double vx_NW_point){ //d(vx*vy)/dx
	double aws;
	aws=((vx_C_point+vx_N_point)*(vy_C_point+vy_E_point) - (vx_W_point+vx_NW_point)*(vy_W_point+vy_C_point))/2*delx;
	aws+=(gam/2*delx)*(d_absf(vx_C_point+vx_N_point)*(vy_C_point-vy_E_point) - d_absf(vx_W_point+vx_NW_point)*(vy_W_point-vy_C_point));
	return aws;
}

__device__ double del_sqr_vy_del_sqr_x(double delx,double vy_E_point,double vy_C_point,double vy_W_point){//d²(vy)/dx²
	return (vy_E_point-2*vy_C_point+vy_W_point)/SQR(delx);
}

__device__ double del_sqr_vy_del_sqr_y(double dely,double vy_N_point,double vy_C_point,double vy_S_point){//d²(vy)/dy²
	return (vy_N_point-2*vy_C_point+vy_S_point)/SQR(dely);
}

__global__ void comp_FG(int imax,int jmax,double gam,double delx,double dely,double Re, double gx,double gy,double del_time, double *d_vx,double *d_vy,double *d_F,double *d_G){

	int idx = blockIdx.x * blockDim.x + threadIdx.x;	
	int n = (imax+2)*(jmax+2);
	
	if(idx < n && idx > (imax+1) && idx < (jmax+1)*(imax+2) && idx%(imax+2) > 0 && idx%(imax+2) < (imax)){
		double term1=del_sqr_vx_del_sqr_x(delx,d_vx[idx+1],d_vx[idx],d_vx[idx-1]); //del_sqr_vx_del_sqr_x(matrix[j][i+1].vx, matrix[j][i].vx,matrix[j][i-1].vx);
				
		double term2=del_sqr_vx_del_sqr_y(dely,d_vx[idx-(imax+2)],d_vx[idx],d_vx[idx+(imax+2)]); //del_sqr_vx_del_sqr_y(matrix[j-1][i].vx, matrix[j][i].vx,matrix[j+1][i].vx);
				
		double term3=del_vx_sqr_del_x(gam,delx,d_vx[idx],d_vx[idx-1],d_vx[idx+1]); //del_vx_sqr_del_x(matrix[j][i].vx,matrix[j][i-1].vx,matrix[j][i+1].vx);
				
		double term4=del_vx_vy_del_y(gam,dely,d_vx[idx-(imax+2)],d_vx[idx],d_vx[idx+(imax+2)],d_vy[idx],d_vy[idx+1],d_vy[idx+(imax+2)],d_vy[idx+(imax+2)+1]); 
		//del_vx_vy_del_y(matrix[j-1][i].vx,matrix[j][i].vx,matrix[j+1][i].vx, matrix[j][i].vy,matrix[j][i+1].vy,matrix[j+1][i].vy,matrix[j+1][i+1].vy);
		
		d_F[idx] = d_vx[idx] + del_time*(((term1+term2)/Re) - term3 - term4 + gx); //matrix[j][i].F=matrix[j][i].vx + del_time*(((term1+term2)/Re) - term3 - term4 + gx);	
	
	}
	else if(idx < n && idx > (imax+1) &&idx < (jmax+1)*(imax+2) && (idx%(imax+2)==0 || idx%(imax+2)==(imax+1)) ) {
		d_F[idx] = d_vx[idx]; //matrix[j][imax].F=matrix[j][imax].vx;
	}
	
	if(idx < n && idx >= 2*(imax+2) && idx < (jmax+1)*(imax+2) && idx%(imax+2) > 0 && idx%(imax+2) < (imax+1)){
		
		//del_sqr_vy_del_sqr_x(matrix[j][i+1].vy,matrix[j][i].vy,matrix[j][i-1].vy);
		double term5=del_sqr_vy_del_sqr_x(delx,d_vy[idx+1],d_vy[idx],d_vy[idx-1]);		
		
		//del_sqr_vy_del_sqr_y(matrix[j-1][i].vy,matrix[j][i].vy,matrix[j+1][i].vy);
		double term6=del_sqr_vy_del_sqr_y(dely,d_vy[idx-(imax+2)],d_vy[idx],d_vy[idx+(imax+2)]); 
		
		//del_vy_vx_del_x(matrix[j][i-1].vy, matrix[j][i].vy,matrix[j][i+1].vy, matrix[j-1][i].vx,matrix[j][i].vx, matrix[j][i-1].vx ,matrix[j-1][i-1].vx);
		double term7=del_vy_vx_del_x(gam,delx,d_vy[idx-1],d_vy[idx],d_vy[idx+1],d_vx[idx-(imax+2)],d_vx[idx],d_vx[idx-1],d_vx[idx-(imax+2)-1]); 
				
		//del_vy_sqr_del_y(matrix[j-1][i].vy ,matrix[j][i].vy,matrix[j+1][i].vy);
		double term8=del_vy_sqr_del_y(gam,dely,d_vy[idx-(imax+2)],d_vy[idx],d_vy[idx+(imax+2)]);
		
		d_G[idx]=d_vy[idx] + del_time*(((term5+term6)/Re) - term7 - term8 + gy);
	
	}
	else if(idx < n && idx > (imax+1) && idx%(imax+2) > 0 && idx%(imax+2) < (imax+1)){
		d_G[idx] = d_vy[idx];
	}
	
	
}


__global__ void comp_RHS(int imax,int jmax,double delx,double dely,double del_time, double *d_rhs,double *d_F,double *d_G){
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	
	int n = (imax+2)*(jmax+2);
	
	if(idx < n && idx > (imax+1) && idx < (jmax+1)*(imax+2) && idx%(imax+2) > 0 && idx%(imax+2) < (imax+1)){
		d_rhs[idx]=((d_F[idx]-d_F[idx-1])/delx +(d_G[idx]-d_G[idx+(imax+2)])/dely)/del_time;
	}	
}


__global__ void build_poisson_system(int jmax,int imax,double delx,double dely,double *d_diag_n, double *d_diag_s, double *d_diag_e,double *d_diag_w,double *d_diag_p){
	int i,j,idx;
	int ew,ee,es,en;
	for(j=jmax;j>0;j--){
		es=j<jmax?1:0;
		en=j>1?1:0;
		for(i=1;i<imax+1;i++){
			idx=j*(imax+2)+i;
			ew=i>1?1:0;
			ee=i<imax?1:0;
			d_diag_e[idx]=ee/SQR(delx);
			d_diag_w[idx]=ew/SQR(delx);
			d_diag_p[idx]=((ee+ew)/SQR(delx) + (en+es)/SQR(dely));
			d_diag_n[idx]=en/SQR(dely);
			d_diag_s[idx]=es/SQR(dely);			
		}
	}

}

__global__ void reductionMax(int imax,int jmax, double * d_partial, double * d_diff,double * d_flag){
	
	__shared__ 	double cache [REDCT_NUMTHREADS];
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	double temp = 0;
	int n = (imax+2)*(jmax+2);
	int inc = blockDim.x*gridDim.x;
	int i,k;
	
	for(i=idx;i<n;i+=inc){
		if(d_flag[i]){
			if(temp < d_diff[i]) temp = d_diff[i];
		}
	}
	
	cache[threadIdx.x]=temp;
	__syncthreads();
	
	for( k =(blockDim.x >> 1);k>0;k>>=1){
		if(threadIdx.x < k){
			if(cache[threadIdx.x] < cache[threadIdx.x+k]){
				cache[threadIdx.x] = cache[threadIdx.x+k];
			}
		}
		__syncthreads();
	}	
	if(threadIdx.x==0){
		d_partial[blockIdx.x]=cache[0];
	}

}

__global__ void red_SOR(int imax,int jmax,double omg, double *d_p, double * d_rhs,double *d_p_diff ,double * d_diag_n,double * d_diag_s,double * d_diag_e,double * d_diag_w,double * d_diag_p,double *d_p_prev,double *d_flag){
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int size=(imax+2)*(jmax+2);
	int line = idx / (imax+2);
	int paridade = line % 2;
	double aux;
	
	if(idx < size && idx > (imax+1) && idx < (imax+2)*(jmax+1) && idx % 2 == paridade && idx % (imax+2) > 0 && idx % (imax+2) < (imax+1)){
		d_p_prev[idx]=d_p[idx];
		aux = d_diag_s[idx]*d_p[idx+(imax+2)] + d_diag_n[idx]*d_p[idx-(imax+2)]+ d_diag_e[idx]*d_p[idx+1] + d_diag_w[idx]*d_p[idx-1];
		aux=(1-omg)*d_p[idx] + omg*(aux-d_rhs[idx])/d_diag_p[idx];
		d_p_diff[idx]=d_absf(aux-d_p[idx]);		
		d_flag[idx]=1;
		d_p[idx]=aux;
	}
	else{
		d_flag[idx]=0;
	}
	
}

__global__ void black_SOR(int imax,int jmax,double omg, double *d_p, double * d_rhs,double *d_p_diff ,double * d_diag_n,double * d_diag_s,double * d_diag_e,double * d_diag_w,double * d_diag_p,double *d_p_prev,double *d_flag){
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int size=(imax+2)*(jmax+2);
	int line = idx / (imax+2);
	int paridade = !(line % 2);
	double aux;
		
	if(idx < size  && idx > (imax+1) && idx < (imax+2)*(jmax+1) && idx % 2 == paridade && idx % (imax+2) > 0 && idx % (imax+2) < (imax+1)){
		d_p_prev[idx]=d_p[idx];
		aux = d_diag_s[idx]*d_p[idx+(imax+2)] + d_diag_n[idx]*d_p[idx-(imax+2)]+d_diag_e[idx]*d_p[idx+1]+d_diag_w[idx]*d_p[idx-1];
		aux=(1-omg)*d_p[idx] + omg*(aux-d_rhs[idx])/d_diag_p[idx];
		d_p_diff[idx]=d_absf(aux-d_p[idx]);		
		d_flag[idx]=1;
		d_p[idx]=aux;
	}
	else{
		d_flag[idx]=0;
	}	
}



int Poisson(){
	int iter=0;
	int i;
	double diff=0;
	double partial[NUMBLOCKS];
	
	while(iter < max_iter){
		for(i=0;i<n_iter;i++){
			red_SOR<<<n_blocos,n_threads>>>(imax,jmax,omg, d_p,d_rhs,d_p_diff ,d_diag_n,d_diag_s,d_diag_e,d_diag_w,d_diag_p,d_p_prev,d_flag);
			black_SOR<<<n_blocos,n_threads>>>(imax,jmax,omg, d_p,d_rhs,d_p_diff ,d_diag_n,d_diag_s,d_diag_e,d_diag_w,d_diag_p,d_p_prev,d_flag);
		}
		
		diff=0;
		reductionMax<<<REDCT_NUMTHREADS,NUMBLOCKS>>>(imax,jmax,d_partial,d_p_diff,d_flag);
		err = hipMemcpy(partial, d_partial, NUMBLOCKS*sizeof(double), hipMemcpyDeviceToHost);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to copy pointer d_partial from device to host (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}
		
		for(i=0;i<NUMBLOCKS;i++){
			if(partial[i] > diff) diff=partial[i];
		}
		iter+=n_iter;
		
		if(diff < eps){
			return iter;
		}	
	}
	return iter;
}


__global__ void d_adap_Vel(int imax,int jmax,double delx,double dely,double del_time, double * d_vx,double *d_vy,double *d_p,double *d_F,double *d_G, double *d_vxdiff,double *d_vydiff,double *d_vxflag,double *d_vyflag){

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int n=(imax+2)*(jmax+2);
	
	
	double aux;
	
	if(idx < n && idx > (imax+1) && idx < (jmax+1)*(imax+2) && idx%(imax+2) > 0 && idx%(imax+2) < (imax)){
		aux=d_F[idx] - (del_time*(d_p[idx+1] - d_p[idx])/delx); //matrix[j][i].F - (del_time*(matrix[j][i+1].p - matrix[j][i].p)/delx);
		d_vxdiff[idx]=d_absf(aux-d_vx[idx]);   //absf(aux-matrix[j][i].vx);
		d_vxflag[idx]=1;
		d_vx[idx]=aux;		
	}
	else{
		d_vxflag[idx]=0;
	}
	
	if(idx < n && idx >= 2*(imax+2) && idx < (jmax+1)*(imax+2) && idx%(imax+2) > 0 && idx%(imax+2) < (imax+1)){
		aux=d_G[idx] - (del_time*(d_p[idx-(imax+2)] - d_p[idx])/dely); //matrix[j][i].G - (del_time*(matrix[j-1][i].p - matrix[j][i].p)/dely);
		d_vydiff[idx]=d_absf(aux-d_vy[idx]);//absf(aux-matrix[j][i].vy);
		d_vyflag[idx]=1;
		d_vy[idx]=aux;
	}
	else{
		d_vyflag[idx]=0;
	}	
	
}

int adap_Vel(int n_blocos, int n_threads){
	
	double diffvx=0;
	double diffvy=0;
	int i;
	
	double partialvx[NUMBLOCKS];
	double partialvy[NUMBLOCKS];
	
	
	d_adap_Vel<<< n_blocos, n_threads >>>(imax,jmax,delx,dely,del_time,d_vx,d_vy,d_p,d_F,d_G,d_vxdiff,d_vydiff,d_vxflag,d_vyflag);		
	reductionMax<<<REDCT_NUMTHREADS,NUMBLOCKS>>>(imax,jmax,d_partial,d_vxdiff,d_vxflag);
	err = hipMemcpy(partialvx, d_partial, NUMBLOCKS*sizeof(double), hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy pointer d_partial from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	reductionMax<<<REDCT_NUMTHREADS,NUMBLOCKS>>>(imax,jmax,d_partial,d_vydiff,d_vyflag);
	err = hipMemcpy(partialvy, d_partial, NUMBLOCKS*sizeof(double), hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy pointer d_partial from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
		
	for(i=0;i<NUMBLOCKS;i++){
		if(partialvy[i] > diffvy) diffvy=partialvy[i];
		if(partialvx[i] > diffvx) diffvx=partialvx[i];
	}
	if(diffvy < eps && diffvx < eps) return 1;
	return 0;
}


int main(int argc, char ** argv)
{
	read_file(argv[1]);
    alocate_vectors_host();
    alocate_vectors_device();
	copy_vectors_host_to_device();
    
    n_threads = (imax+2);
    n_blocos = ((imax+2)*(jmax+2)+ n_threads-1)/n_threads;
    init_UVP<<< n_blocos, n_threads >>>(imax,jmax,vx_init,vy_init,p_init,d_vx,d_vy,d_p,d_rhs,d_F,d_G);
	build_poisson_system<<<1,1>>>(jmax,imax,delx,dely,d_diag_n,d_diag_s,d_diag_e,d_diag_w,d_diag_p);
	
	int set_time=1;
	double ant_del_time=1.0;
	double eps_time=1e-7;
	int num_time=0;
	int limit=100;
    gettimeofday(&start, NULL);
	while(!state){
		if(set_time){
			comp_delt();
		}
		if(absf(ant_del_time-del_time) < eps_time){
			num_time++;
			if(num_time==limit){
				set_time=0;
			}
		}
		else{
			num_time=0;
		}
		
		set_bondCond();		
		set_lidDrivenCavityProblem<<< n_blocos,n_threads>>>(1.0,imax,jmax,d_vx);
		
		comp_FG<<< n_blocos,n_threads >>>(imax,jmax,gam,delx,dely,Re,gx,gy,del_time,d_vx,d_vy,d_F,d_G);		
		comp_RHS<<< n_blocos,n_threads >>>(imax,jmax,delx,dely,del_time,d_rhs,d_F,d_G);		
		Poisson();
		state=adap_Vel(n_blocos,n_threads);
		
		ttime+=del_time;
		ant_del_time=del_time;		
	}
	set_bondCond();		
	set_lidDrivenCavityProblem<<< n_blocos,n_threads>>>(1.0,imax,jmax,d_vx);
	gettimeofday(&end, NULL);

	copy_vectors_device_to_host(); 
    write_file(argv[2]);
    
    
    free_vectors_device();	
	free_vectors_host();
	
	return 0;
}

